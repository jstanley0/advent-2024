#include "hip/hip_runtime.h"
﻿
#include <iostream>
#include <fstream>
#include <vector>
#include <windows.h>

constexpr size_t PRICE_TABLE_SIZE = 2000;

__global__ void hash(uint32_t *secrets, uint32_t secrets_sz, int8_t *price_table_base) {
    unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= secrets_sz)
        return;
    int8_t* start = price_table_base + idx * PRICE_TABLE_SIZE;
    int8_t* end = start + PRICE_TABLE_SIZE;
    uint32_t n = secrets[idx];
    for (int8_t* i = start; i < end; ++i) {
        n = ((n << 6) ^ n) & 0xFFFFFF;
        n = ((n >> 5) ^ n);
        n = ((n << 11) ^ n) & 0xFFFFFF;
        *i = n % 10;
    }
}

__global__ void search(int8_t *price_table_base, uint32_t secrets_sz, int32_t *best) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n > 19 * 19 * 19 * 19)
        return;

    int d0 = (n / (19 * 19 * 19)) - 9;
    int d1 = ((n / (19 * 19)) % 19) - 9;
    int d2 = ((n / 19) % 19) - 9;
    int d3 = (n % 19) - 9;
    
    int32_t bid = 0;
    int8_t* prices = price_table_base;
    for (uint32_t m = 0; m < secrets_sz; ++m) {
        for (int i = 1; i < PRICE_TABLE_SIZE - 4; ++i) {
            if (prices[i] - prices[i - 1] == d0 &&
                prices[i + 1] - prices[i] == d1 &&
                prices[i + 2] - prices[i + 1] == d2 &&
                prices[i + 3] - prices[i + 2] == d3)
            {
                bid += prices[i + 3];
                break;
            }
        }
        prices += PRICE_TABLE_SIZE;
    }
    atomicMax(best, bid);
}

int main(int argc, char **argv)
{
    if (argc < 2) {
        std::cerr << "usage: " << argv[0] << " infile" << std::endl;
        return 1;
    }

    LARGE_INTEGER before;
    QueryPerformanceCounter(&before);

    std::ifstream infile(argv[1]);
    std::vector<uint32_t> secrets;
    uint32_t n;
    while (infile >> n) {
        secrets.push_back(n);
    }

    uint32_t* deviceSecrets;
    hipMalloc(&deviceSecrets, secrets.size() * sizeof(uint32_t));
    hipMemcpy(deviceSecrets, secrets.data(), secrets.size() * sizeof(uint32_t), hipMemcpyHostToDevice);

    int8_t* devicePriceTable;
    hipMalloc(&devicePriceTable, secrets.size() * PRICE_TABLE_SIZE * sizeof(int8_t));
    hash<<<(secrets.size() + 255) / 256, 256>>>(deviceSecrets, secrets.size(), devicePriceTable);

    int32_t* deviceBest;
    hipMalloc(&deviceBest, sizeof(int32_t));
    hipMemset(deviceBest, 0, sizeof(int32_t));
    search<<<510, 256>>>(devicePriceTable, secrets.size(), deviceBest);
    int32_t best;
    hipMemcpy(&best, deviceBest, sizeof(int32_t), hipMemcpyDeviceToHost);
    std::cout << best << std::endl;

    hipFree(deviceBest);
    hipFree(devicePriceTable);
    hipFree(deviceSecrets);

    LARGE_INTEGER after;
    LARGE_INTEGER frequency;
    QueryPerformanceCounter(&after);
    QueryPerformanceFrequency(&frequency);
    printf("Elapsed time: %.3f seconds\n", (double)(after.QuadPart - before.QuadPart) / frequency.QuadPart);

    return 0;
}
