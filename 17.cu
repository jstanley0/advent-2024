#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <windows.h>

#define CHECKS_PER_THREAD 16384ULL
#define THREADS_PER_BLOCK 1024
#define BLOCKS 256

__device__ bool check(uint64_t a) {
    uint64_t b, r = 0;
    while (a != 0) {
        b = (a & 7) ^ 1;
        b ^= (a >> b) ^ 4;
        a >>= 3;
        r = (r << 3) | b & 7;
    }
    return r == 02411754603145530ULL;
}

__global__ void search(uint64_t batch_start, uint64_t* result) {
    unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;

    uint64_t range_start = batch_start + idx * CHECKS_PER_THREAD;
    uint64_t range_end = range_start + CHECKS_PER_THREAD;

    for (uint64_t i = range_start; i < range_end; ++i) {
        if (check(i)) {
            atomicMin(result, i);
            break;
        }
    }
}

__managed__ uint64_t result = ULLONG_MAX;

int main()
{
    LARGE_INTEGER before;
    QueryPerformanceCounter(&before);

    uint64_t a = 201972175280682ULL;
    for(;;) {
        search<<<BLOCKS, THREADS_PER_BLOCK>>>(a, &result);
        hipDeviceSynchronize();
        if (result != ULLONG_MAX)
            break;

        a += BLOCKS * THREADS_PER_BLOCK * CHECKS_PER_THREAD;
        putchar('.');
    } 
    printf("\n%llx (%lld)\n", result, result);

    LARGE_INTEGER after;
    LARGE_INTEGER frequency;
    QueryPerformanceCounter(&after);
    QueryPerformanceFrequency(&frequency);
    printf("Elapsed time: %.3f seconds\n", (double)(after.QuadPart - before.QuadPart) / frequency.QuadPart);

    return 0;
}
